#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

// argv[1]: malloc size;
// argv[2]: gpuidx
int main(int argc, char **argv)
{
    hipError_t err = hipSuccess;
    //size_t size = 0x10000000;//256M
    void *dPtr = NULL;
    void *dPtr2 = NULL;

    if (argc < 3) {
        printf("paramter error\n");
        return -1;
    }
    
    size_t size = strtol(argv[1], NULL, 16);
    int gpuidx = atoi(argv[2]);
    printf("try to malloc %lx from GPU:%d\n", size, gpuidx);


//    printf("size_t len:%lu\n", sizeof(size_t)); //8 bytes
    printf("try to init after any key\n");getchar();
    hipSetDevice(gpuidx);
    printf("try to init gpu:%d after any key\n", gpuidx);getchar();
    hipFree(0);
    printf("try to malloc size:%lx, on gpu:%d after any key\n", size, gpuidx);getchar();
    err = hipMalloc(&dPtr, size);
    if (err != hipSuccess)
    {
        printf("Failed to allocate cuda memory: err:%d, %s!\n", (int)err, hipGetErrorString(err));
        return -1;
    }
    size *= 2;
    printf("try to malloc 2 size:%lx, on gpu:%d after any key\n", size, gpuidx);getchar();
    err = hipMalloc(&dPtr2, size);
    if (err != hipSuccess)
    {
        printf("Failed to allocate cuda memory: err:%d, %s!\n", (int)err, hipGetErrorString(err));
        return -1;
    }
    hipFree(dPtr);
    hipFree(dPtr2);
    //sleep(200000);

    return 0;
}
