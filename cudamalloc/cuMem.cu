#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{

    hipError_t err = hipSuccess;
    int size = 1024*1024*1024;
    void *dPtr = NULL;

    err = hipMalloc(&dPtr, size);

    if (err != hipSuccess)
    {
        printf("Failed to allocate cuda memory!\n");
        return -1;
    }

    printf("Allocated cuda memory successfully\n");

    err = hipMemset(dPtr, 0x0, size);

//    cudaFree(dPtr);

    //cudaFree(0);
    return 0;
}
