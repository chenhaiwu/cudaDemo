#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

int main(void)
{

    hipError_t err = hipSuccess;
    int size = 10*1024*1024;
    void *dPtr = NULL;


    for (int i = 0; i < 2; i ++) {
        hipSetDevice(i);
        printf("Try to allocate cuda memory 10M\n");
        err = hipMalloc(&dPtr, size);

        if (err != hipSuccess)
        {
            printf("Failed to allocate cuda memory: err:%d, %s!\n", (int)err, hipGetErrorString(err));
            return -1;
        }

        printf("Allocated cuda memory successfully\n");

        err = hipMemset(dPtr, 0x0, size);

        hipFree(dPtr);
    }

    sleep(20000);

    return 0;
}
