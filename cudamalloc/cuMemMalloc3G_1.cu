#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

int main(void)
{

    hipError_t err = hipSuccess;
    size_t size = 0xcc000000;//3G
    void *dPtr = NULL;

//    printf("size_t len:%lu\n", sizeof(size_t)); //8 bytes
    hipFree(0);
    err = hipMalloc(&dPtr, size);

    if (err != hipSuccess)
    {
        printf("Failed to allocate cuda memory!\n");
        return -1;
    }

    printf("Allocated cuda 1G memory successfully, ptr=0x%016llx\n", dPtr);
    sleep(20);

    return 0;
}
