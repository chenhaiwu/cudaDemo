#include <builtin_types.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>


#include <cstring>
#include <iostream>
#include <string>


hipDevice_t cuDevice_0;
hipDevice_t cuDevice_1;

hipCtx_t cuContext_0;
hipCtx_t cuContext_1;

inline void checkCudaErrors(hipError_t result)
{
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString((hipError_t)result));
		exit(EXIT_FAILURE);
	}
}

void printhex(void *p, int len)
{
	int i = 0;
	unsigned long long *pp = (unsigned long long *)p;
	printf("==============haiwu dump addr:0x%016x===================\n", pp);
	for (i = 0; i < len / 8; i++) {
		if (i % 2 == 0) {
			printf("0x%08x\t\t", i * 8);
		}
		printf("%016x  ", (unsigned int)(*(pp + i)));
		if (i % 2 == 1) {
			printf("\n");
		}
	}
	printf("\n");
}


static hipError_t initCUDA(int dev_id) {
	hipFunction_t cuFunction = 0;
	hipError_t status;
	int major = 0, minor = 0;
	char deviceName[100];
	std::string module_path, ptx_source;
	hipDevice_t cuDevice = dev_id == 0 ? cuDevice_0 : cuDevice_1;
	hipCtx_t cuContext = dev_id == 0 ? cuContext_0 : cuContext_1;
	int ctx_size = sizeof(cuContext);


	checkCudaErrors(hipDeviceGet(&cuDevice, dev_id));
	hipDeviceGetName(deviceName, 100, cuDevice);
	printf("> Using CUDA Device [%d]: %s\n", dev_id, deviceName);

	// get compute capabilities and the devicename
	checkCudaErrors(hipDeviceGetAttribute(
	  &major, hipDeviceAttributeComputeCapabilityMajor, cuDevice));
	checkCudaErrors(hipDeviceGetAttribute(
	  &minor, hipDeviceAttributeComputeCapabilityMinor, cuDevice));
	checkCudaErrors(hipDeviceGetName(deviceName, sizeof(deviceName), cuDevice));
	printf("> GPU Device has SM %d.%d compute capability\n", major, minor);

	status = hipCtxCreate(&cuContext, 0, cuDevice);
	printf("haiwu================2.5 hipCtxCreate done\n");
	printf("ctx size: %d, print conext 0x2900:\n", ctx_size
);
//	printf("sizeof(hipCtx_t):%d, sizeof(hipCtx_t):%d\n", p_ctx_size, ctx_size);
	printhex(cuContext, 0x2900);

	return hipSuccess;
Error:
  hipCtxDestroy(cuContext);
  return status;
}


int main()
{
	printf("create context on device 0 start\n");
	hipError_t err = hipInit(0);
	if (hipSuccess != err) {
		printf("hipInit fail\n");
	}
	initCUDA(0);
	printf("create context on device 0 end\n");

	sleep(5);

	printf("create context on device 1 start\n");
	initCUDA(1);
	printf("create context on device 1 done\n");
	sleep(5);

	return 0;
}

/*
int main(int argc, char const *argv[])
{
    if (InitCUDA()) {
        printf("CUDA initialized.\n");
    }
 
    return 0;
}
*/

