#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <assert.h>
#include <errno.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>

inline void checkCudaErrors(hipError_t result)
{
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
		exit(EXIT_FAILURE);
    }
}


void printDeviceProp(const hipDeviceProp_t &prop)//hipPointerAttribute_t
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %lu.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %lu.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %lu.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %lu.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %lu.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
    printf("unifiedAddressing : %d.\n", prop.unifiedAddressing);
}
 
bool InitCUDA()
{
    //used to count the device numbers
    int count;
 
    // get the cuda device count
    hipGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
 
    // find the device >= 1.X
    int i;
    for (i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                printDeviceProp(prop);
                break;
            }
        }
    }
 
    // if can't find the device
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

	sleep(10);

    // set cuda device
    hipSetDevice(i);

	sleep(10);

    return true;
}

int main_v1(int argc, char const *argv[])
{
    if (InitCUDA()) {
        printf("CUDA initialized.\n");
    }
 
    return 0;
}


// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}
            
// Host code
int main_v2()
{
    int N = 100000000;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize input vectors
	printf("set device 0\n");
    hipSetDevice(0);
	printf("set device 0 done\n");

    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =
            (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("run on device 0 done\n");
	sleep(3);

	printf("try to run on devcie 1\n");

    // Initialize input vectors
    hipSetDevice(1);
	printf("set device 1 done\n");

    // Allocate vectors in device memory
    float* d_A_1;
    hipMalloc(&d_A_1, size);
    float* d_B_1;
    hipMalloc(&d_B_1, size);
    float* d_C_1;
    hipMalloc(&d_C_1, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A_1, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B_1, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A_1, d_B_1, d_C_1, N);

	printf("run on device 1 done\n");
	sleep(3);

    // Free device memory
    hipFree(d_A_1);
    hipFree(d_B_1);
    hipFree(d_C_1);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
            
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

	printf("done\n");
}

int main_cudafree()
{
	hipFree(0);
	sleep(5);
	printf("run on device 1 start\n");
	hipSetDevice(1);
	printf("run on device 1 done\n");
	sleep(5);
	printf("run on device 1 free start\n");
	hipFree(0);
	printf("run on device 1 free end\n");
	sleep(5);
	printf("run on device 0 start\n");
	hipSetDevice(0);
	printf("run on device 0 done\n");
	sleep(5);
	printf("run on device 0 hipMalloc start\n");
    float* d_A_0;
//    hipMalloc(&d_A_0, 100);
	size_t v_free, v_total;
//	hipMemGetInfo(&v_free, &v_total);
	printf("run on device 0, cudamemgetinfo:free:%lu, total:%lu\n", v_free, v_total);
//	printf("run on device 0 hipMalloc end\n");
	sleep(5);
	printf("run on device 1 start\n");
	hipSetDevice(1);
	printf("run on device 1 done\n");
	sleep(5);
	printf("run on device 1 hipMalloc start\n");
    float* d_A_1;
    //hipMalloc(&d_A_1, 100);
	hipMemGetInfo(&v_free, &v_total);
	printf("run on device 1, cudamemgetinfo:free:%lu, total:%lu\n", v_free, v_total);
	//printf("run on device 1 hipMalloc end\n");
	return 0;
}

void print_array(int* array, int size) {
	int i;
	for (i = 0; i < size; i++) {
	    printf("%x ", array[i]);
	}
	printf("\n");
}

__global__ void VecAdd2(int* A, int* B, int* C) {
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

int main_v3()
{
//    int N = 0x10000000;
    int N = 4;

    int size = N * sizeof(int);
	
    int* h_A = (int*)malloc(size);
    int* h_B = (int*)malloc(size);
    int* h_C = (int*)malloc(size);
	memset(h_A, 0x11111111, size);
	memset(h_B, 0x22222222, size);
	int all_card = 1;
	
	for (int i = 0; i < 8 && all_card; i++) {
		size_t v_free, v_total;
		printf("hipSetDevice on device %d start, any key to continue\n", i);getchar();
		hipSetDevice(i);
		printf("======hipMalloc on device %d end, any key to continue\n", i);getchar();
		int* d_A;
		hipMalloc(&d_A, size);
	    int* d_B;
	    hipMalloc(&d_B, size);
	    int* d_C;
	    hipMalloc(&d_C, size);
		printf("======hipMemcpy on gpu:%d, any key to continue\n", i);getchar();

		// Copy vectors from host memory to device memory
		hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
		hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
		memset(h_C, 0x0, size);
		printf("======VecAdd run kernel on gpu:%d, any key to continue\n", i);getchar();

		// Invoke kernel
		int threadsPerBlock = 256;
		int blocksPerGrid =
		        (N + threadsPerBlock - 1) / threadsPerBlock;
//		VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
		VecAdd2<<<1, N>>>(d_A, d_B, d_C);

		printf("======hipMemcpy run kernel on gpu:%d, any key to continue\n", i);getchar();
		// h_C contains the result in host memory
		hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		for (int j = 0; j < size / 4; j++) {
			if (h_C[j] != 0x33333333) {
				printf("NOT espect value, :%f\n", h_C[j]);
				all_card = 0;
				print_array(h_C, N);
				break;
			}
		}

//		hipMemGetInfo(&v_free, &v_total);
		printf("======hipFree run kernel on gpu:%d, any key to continue\n", i);getchar();
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
	}

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);


	return 0;
}


int main_v4()
{
	size_t size = 0x21000000;
	
    int* h_A = (int*)malloc(size);
	memset(h_A, 0x11111111, size);
	int all_card = 1;
	
	for (int i = 0; i < 8 && all_card; i++) {
		printf("hipSetDevice on device %d start, any key to continue\n", i);getchar();
		hipSetDevice(i);
		printf("======hipFree on device %d, any key to continue\n", i);getchar();
		hipFree(0);
		printf("======hipMalloc on device %d, any key to continue\n", i);getchar();
		int* d_A;
		hipMalloc(&d_A, size);
		printf("======hipMemcpy on gpu:%d, any key to continue\n", i);getchar();

		// Copy vectors from host memory to device memory
		hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

		printf("======hipFree run kernel on gpu:%d, any key to continue\n", i);getchar();
		hipFree(d_A);
	}

    // Free host memory
    free(h_A);

	return 0;
}

int main_v5()
{
	int N = 0x10000;
	int size = N * sizeof(int);
	
    int* h_A = (int*)malloc(size);
	memset(h_A, 0x11111111, size);
	int all_card = 1;
	
	for (int i = 0; i < 8 && all_card; i++) {
		printf("hipSetDevice on device %d start, any key to continue\n", i);getchar();
		hipSetDevice(i);
		printf("======hipMalloc on device %d end, any key to continue\n", i);getchar();
		int* d_A;
		hipMalloc(&d_A, size);
		printf("======hipMemcpy on gpu:%d, any key to continue\n", i);getchar();

//		hipPointerAttribute_t *cuda_p_attr = malloc(hipPointerAttribute_t);
		hipPointerAttribute_t cuda_p_attr;
		
		hipPointerGetAttributes(&cuda_p_attr, d_A);
		printf("d_A:0x%p, from device:%d, host_p:0x%p, device_p:0x%p, type:%d\n",
//			d_A, cuda_p_attr->device, cuda_p_attr->hostPointer, cuda_p_attr->devicePointer, (int)(cuda_p_attr->type));
			d_A, cuda_p_attr.device, cuda_p_attr.hostPointer, cuda_p_attr.devicePointer, (int)(cuda_p_attr.type));


		// Copy vectors from host memory to device memory
		hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

		printf("======hipFree run kernel on gpu:%d, any key to continue\n", i);getchar();
		hipFree(d_A);
	}

    // Free host memory
    free(h_A);

	return 0;
}


int main(int argc, char const *argv[])
{
	main_v4();
    return 0;
}


