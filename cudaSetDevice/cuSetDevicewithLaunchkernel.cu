#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
 
void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %lu.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %lu.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %lu.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %lu.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %lu.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}
 
bool InitCUDA()
{
    //used to count the device numbers
    int count;
 
    // get the cuda device count
    hipGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
 
    // find the device >= 1.X
    int i;
    for (i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                printDeviceProp(prop);
                break;
            }
        }
    }
 
    // if can't find the device
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

	size_t size = 1024 * sizeof(float);
	hipSetDevice(0);            // Set device 0 as current
	float* p0;
	hipMalloc(&p0, size);       // Allocate memory on device 0
	MyKernel<<<1000, 128>>>(p0); // Launch kernel on device 0
	hipSetDevice(1);            // Set device 1 as current
	float* p1;
	hipMalloc(&p1, size);       // Allocate memory on device 1
	MyKernel<<<1000, 128>>>(p1); // Launch kernel on device 1

	sleep(10);

    // set cuda device
    hipSetDevice(i);

	sleep(10);

    return true;
}
 
int main(int argc, char const *argv[])
{
    if (InitCUDA()) {
        printf("CUDA initialized.\n");
    }
 
    return 0;
}
